
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N){

    printf("Caculating On GPU\n");
    for (int idx=0; idx<N; idx++){
        C[idx] = A[idx] + B[idx];
    }
}


void initialData(float *ip, int size){
    // generate different seed for random number
    time_t t;
    srand((unsigned int) time(&t));

    for (int i=0; i<size; i++){
        ip[i] = (float)(rand() & 0xFF)/10.0f;
    }
}


void print(float *array, const int N){
    for (int idx=0; idx<N; idx++){
        printf(" %f", array[idx]);
    }
    printf("\n");
}

int main(){
    int nElem = 4;
    size_t nBytes = nElem * sizeof(float);
    float *h_A, *h_B, *h_C;

    printf("malloc memory on Host\n");
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    h_C = (float *)malloc(nBytes);

    printf("initialize data on Host\n");
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    print(h_A, nElem);
    print(h_B, nElem);

    printf("malloc memory on GPU\n");
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);
    
    printf("copying inputs from Host to Device\n");
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    sumArraysOnGPU <<<1, 1>>>(d_A, d_B, d_C, nElem); // 异步计算
    printf("copying output from Device to Host\n");
    hipMemcpy(h_C, d_C, nBytes, hipMemcpyDeviceToHost);
    print(h_C, nElem);

    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}



