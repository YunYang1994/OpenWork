
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) C[idx] = A[idx] + B[idx]; // 检查是否越界
    printf("%f + %f = %f Caculated On GPU: block %d thread %d\n", 
            A[idx], B[idx], C[idx], blockIdx.x, threadIdx.x);
}

void initialData(float *ip, int size){
    // generate different seed for random number
    time_t t;
    srand((unsigned int) time(&t));

    for (int i=0; i<size; i++){
        ip[i] = (float)(rand() & 0xFF)/10.0f;
    }
}

void print(float *array, const int N){
    for (int idx=0; idx<N; idx++){
        printf(" %f", array[idx]);
    }
    printf("\n");
}

int main(){
    int nElem = 6;
    size_t nBytes = nElem * sizeof(float);
    float *h_A, *h_B;

    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);

    initialData(h_A, nElem);
    initialData(h_B, nElem);
    
    printf("向量 A:");
    print(h_A, nElem);
    printf("向量 B:");
    print(h_B, nElem);

    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);
    
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
   
    printf("向量 C 的每个元素计算过程:\n");
    dim3 block(2);
    dim3 thread(3);
    sumArraysOnGPU <<< block, thread >>>(d_A, d_B, d_C, nElem); // 异步计算

    free(h_A);
    free(h_B);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
